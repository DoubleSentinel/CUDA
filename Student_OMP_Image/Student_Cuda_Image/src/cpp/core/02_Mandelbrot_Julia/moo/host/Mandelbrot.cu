
#include <hip/hip_runtime.h>
//#include "Mandelbrot.h"
//
//#include <iostream>
//
//using std::cout;
//using std::endl;
//
///*----------------------------------------------------------------------*\
// |*			Declaration 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Imported	 	*|
// \*-------------------------------------*/
//
//extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h,float t);
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///*----------------------------------------------------------------------*\
// |*			Implementation 					*|
// \*---------------------------------------------------------------------*/
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
//Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt) :
//	Animable_I<uchar4>(grid, w, h, "Mandelbrot_CUDA_rgba_uchar4")
//    {
//    // Input
//    this->dt = dt;  // animation
//
//    // Tools
//    this->t = 0;					// protected dans super classe Animable
//    }
//
//Mandelbrot::~Mandelbrot(void)
//    {
//    // rien
//    }
//
///*--------------------------------------*\
// |*		Public			*|
// \*-------------------------------------*/
//
///**
// * Override
// */
//void Mandelbrot::animationStep()
//    {
//    t += dt;
//    }
//
///*--------------------------------------*\
// |*		Private			*|
// \*-------------------------------------*/
//
///**
// * Override (code naturel omp)
// * Image non zoomable : domaineMath pas use ici
// */
//void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
//    {
//    Device::lastCudaError("mandelbrot rgba uchar4 (before)"); // facultatif, for debug only, remove for release
//
//    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,t);
//
//    Device::lastCudaError("mandelbrot rgba uchar4 (after)"); // facultatif, for debug only, remove for release
//    }
//
///*----------------------------------------------------------------------*\
// |*			End	 					*|
// \*---------------------------------------------------------------------*/
//
