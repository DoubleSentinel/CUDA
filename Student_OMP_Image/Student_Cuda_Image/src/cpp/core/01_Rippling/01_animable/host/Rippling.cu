#include "hip/hip_runtime.h"
#include "Rippling.h"

#include <iostream>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Rippling::Rippling(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Rippling_CUDA_rgba_uchar4")
    {
    // Input
    this->dt = dt;  // animation

    // Tools
    this->t = 0;					// protected dans super classe Animable
    }

Rippling::~Rippling(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Rippling::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Override (code naturel omp)
 * Image non zoomable : domaineMath pas use ici
 */
void Rippling::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    rippling<<<dg,db>>>(ptrDevPixels,w,h,t);

    Device::lastCudaError("rippling rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

