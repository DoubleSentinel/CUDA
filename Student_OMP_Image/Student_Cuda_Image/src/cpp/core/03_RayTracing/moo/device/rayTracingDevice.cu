#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"

#include "IndiceTools_GPU.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		RayTracingDevice.cu et non RayTracing.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int nbspheres, Sphere* ptrDevTabSphere, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(nbspheres, ptrDevTabSphere);

    const int WH=w*h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	rayTracingMath.colorIJ(&ptrDevPixels[s],i, j, t); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

