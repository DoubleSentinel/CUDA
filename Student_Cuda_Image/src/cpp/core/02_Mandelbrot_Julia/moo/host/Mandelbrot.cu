#include "hip/hip_runtime.h"
#include "Mandelbrot.h"
#include "DomaineMath_CPU.h"
#include <iostream>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h,float t, uint n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float t, uint n, const DomaineMath &domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_CUDA_Luy")
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = t;
    }

Mandelbrot::~Mandelbrot(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t += t;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Override (code naturel omp)
 * Image non zoomable : domaineMath pas use ici
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,t,n, domaineMath);

    Device::lastCudaError("mandelbrot rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

