#include "hip/hip_runtime.h"
#include "Mandelbrot.h"
#include "DomaineMath_CPU.h"
#include <iostream>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h,float t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float t, uint n, const DomaineMath &domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_CUDA_Luy", domaineMath),
	variateurAnimation(Interval<uint>(5, 250), t)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;
    }

Mandelbrot::~Mandelbrot(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Override (code naturel omp)
 * Image non zoomable : domaineMath pas use ici
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,this->t,domaineMath);

    Device::lastCudaError("mandelbrot rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

