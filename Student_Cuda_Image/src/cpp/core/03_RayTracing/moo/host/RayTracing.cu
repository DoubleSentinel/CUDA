#include "hip/hip_runtime.h"
#include <iostream>
#include "Device.h"
#include "RayTracing.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(uchar4* ptrDevPixels,uint w, uint h,float t, int nbSphere, Sphere* ptrDevTabSphere);


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_Luy")
    {

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable

    // Tools
    this->nbSphere = 10;
    this->sizeOctet = sizeof(Sphere)*nbSphere;
    SphereCreator sphereCreator = SphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    this->toGM(ptrTabSphere);
    }

RayTracing::~RayTracing()
    {
    //MM free memory
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("RayTracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    raytracing<<<dg,db>>>(ptrDevPixels,w,h,t,this->nbSphere, this->ptrDevTabSphere);

    Device::lastCudaError("RayTracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
void RayTracing::toGM(Sphere* ptrTabSphere)
    {
    // MM (malloc/memclear Device)
    	{
    	Device::malloc(&ptrDevTabSphere, sizeOctet);
    	Device::memclear(ptrDevTabSphere, sizeOctet);
    	}

    // MM (copy Host->Device)
    	{
    	Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);
    	}
    }
void RayTracing::toCM(Sphere* ptrTabSphere)
    {
    return;
    }
void RayTracing::toSM(Sphere* ptrTabSphere)
    {
    return;
    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

