#include "hip/hip_runtime.h"
#include "RayTracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "SphereCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rayTracing(uchar4* ptrDevPixels,int nbspheres, Sphere* ptrDevTabSphere,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_Luy")
    {
    // Time
    this->dt = dt;

    // Inputs
    this->t = 0; // protected dans Animable
    this->nbSphere = 2;
    this->sizeOctet = sizeof(Sphere)*nbSphere;
    SphereCreator sphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();
    // MM
	{
	Device::malloc(&ptrDevTabSphere, sizeOctet);
	Device::memclear(ptrDevTabSphere, sizeOctet);
	Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);
	}
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rayTracing rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    rayTracing<<<dg,db>>>(ptrDevPixels, this->nbSphere, this->ptrDevTabSphere,w,h,t);

    Device::lastCudaError("rayTracing rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

